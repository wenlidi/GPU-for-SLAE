#include "hip/hip_runtime.h"
#include <vector>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <algorithm>
#include<>
#include <numeric>
#include <time.h>
#include "MC_gpu.h"
#include <hiprand/hiprand_kernel.h>

using std::vector;
using std::cout;
using std::endl;
using std::string;
using std::ifstream;
using std::ofstream;
using std::min;

__global__ void initRNG(hiprandState *const rngStates,
	const unsigned int seed)
{
	// Determine thread ID
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Initialise the RNG
	hiprand_init(seed, tid, 0, &rngStates[tid]);
}

__device__ inline void getPoint(float &x, hiprandState &state)
{
	x = hiprand_uniform(&state);
}

__global__ void RunMonteCarlo_kernel(int num_n, int m, int* d_matrix_l, int* d_accu_row_size, float alpha, int* d_out_deg, int* d_w, int id, hiprandState *const rngStates){
	
	unsigned int bid = blockIdx.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid >= m){
		return;
	}

	hiprandState localState = rngStates[tid];

	int current_state = d_matrix_l[d_accu_row_size[id] + (tid % d_out_deg[id])];
	atomicAdd(d_w + current_state, 1);

	
	float x;
	getPoint(x, localState);
	while (x < 0.85 && d_out_deg[current_state] != 0){
		getPoint(x, localState);
		//TODO
		int random_walk = x * d_out_deg[current_state];
		random_walk = random_walk % d_out_deg[current_state];
		current_state = d_matrix_l[d_accu_row_size[current_state] + random_walk];
		atomicAdd(d_w + current_state, 1);

		getPoint(x, localState);
	}
}

void simulateMC(int device_id, int num_n, int* &d_matrix_l, int* &d_accu_row_size, float alpha, int num_m, unsigned int seed, int threads_per_block, int* &out_deg, int* &d_out_deg){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);
	dim3 dim_grid, dim_block;

	dim_block.x = threads_per_block;

	int *d_w;
	hipMalloc((void **)&d_w, num_n * sizeof(int));
	hipMemset(d_w, 0, num_n * sizeof(int));


	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);
	sdkResetTimer(&hTimer);

	for (int i = 0; i < num_n; i++){
		if (out_deg[i] == 0){
			continue;
		}
		//TODO
		int m = num_m * alpha / out_deg[i];
		m = m * out_deg[i];
		//init
		dim_grid.x = m / dim_block.x;
		if (dim_grid.x == 0) dim_grid.x = 1;
		hiprandState *d_rngStates;
		hipMalloc((void **)&d_rngStates, dim_grid.x * dim_block.x * sizeof(hiprandState));

		initRNG << <dim_grid, dim_block >> >(d_rngStates, seed);
		hipDeviceSynchronize();

		sdkStartTimer(&hTimer);
		//cuda
		RunMonteCarlo_kernel << <dim_grid, dim_block>> >(num_n, m, d_matrix_l, d_accu_row_size, alpha, d_out_deg, d_w, i, d_rngStates);
		
		hipDeviceSynchronize();
		sdkStopTimer(&hTimer);
		
		//cout << "Elapsed Time: " << sdkGetTimerValue(&hTimer) << "ms" << endl;
		//
		cout << "finish" << i << endl;
		hipFree(d_rngStates);
	}

	vector<unsigned int> results(num_n);
	hipMemcpy(&results[0], d_w, num_n * sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	/*
	for (int i = 0; i < num_n; i++){
		cout << results[i] << " ";
	}
	cout << endl;
	*/
	float sum_w = 0;
	for (int i = 0; i< num_n; i++){
		sum_w += results[i] * 1.0 / num_m;
	}
	//float sum_w = static_cast<int>(std::accumulate(results.begin(), results.end(), 0)) * 1.0 / num_m;

	cout << "Elapsed Time: " << sdkGetTimerValue(&hTimer) << "ms" << endl << endl;

	float sum_ans = 0;
	//cout << sum_w << endl;
	for (int i = 0; i < num_n; i++){
		//cout << results[i] << endl;
		cout << (results[i] * 1.0 / num_m + 1) * 1.0 / (sum_w + num_n) * num_n << endl;
	}
	
	/*
	hiprandStatus_t curandResult;
	hiprandGenerator_t prng;
	curandResult = hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	curandResult = hiprandSetPseudoRandomGeneratorSeed(prng, seed);

	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);
	sdkResetTimer(&hTimer);

	//don't know why first time to generate random number need a lot of time 
	hiprandGenerateUniform(prng, (float *)d_point, 1);

	float generate_random_number_time;
	sdkStartTimer(&hTimer);
	hiprandGenerateUniform(prng, (float *)d_point, dimension * num_m);
	hipDeviceSynchronize();
	sdkStopTimer(&hTimer);
	generate_random_number_time = sdkGetTimerValue(&hTimer);

	cout << "Elapsed Time of generate random number per variable: " << sdkGetTimerValue(&hTimer) << "ms" << endl;
	sdkResetTimer(&hTimer);

	float last_time = 0;
	float fastest_elapsed_time = 0;
	float slowest_elapsed_time = 0;

	for (int i = 0; i < num_n; i++){
		sdkStartTimer(&hTimer);
		RunMonteCarloGPUForOneVariable_Kernel << <dim_grid, dim_block, dim_block.x * sizeof(float)>> >(d_matrix_p, d_column, d_accu_row_size,
			d_matrix_l_row_sum, f_value, d_point, num_m, dimension, i, d_result);
		reduce_sum_array << <(1, 1, 1), dim_grid, dim_grid.x * sizeof(float) >> >(d_result);
		hipDeviceSynchronize();
		sdkStopTimer(&hTimer);

		if (fastest_elapsed_time == 0 || fastest_elapsed_time > sdkGetTimerValue(&hTimer) - last_time){
			fastest_elapsed_time = sdkGetTimerValue(&hTimer) - last_time;
		}
		if (slowest_elapsed_time == 0 || slowest_elapsed_time < sdkGetTimerValue(&hTimer) - last_time){
			slowest_elapsed_time = sdkGetTimerValue(&hTimer) - last_time;
		}
		last_time = sdkGetTimerValue(&hTimer);
		//cout << "Elapsed Time: " << sdkGetTimerValue(&hTimer) << "ms" << endl;
		hipMemcpy(&h_exp[i], d_result, sizeof(float), hipMemcpyDeviceToHost);
		//h_exp[i] = static_cast<float>(std::accumulate(results.begin(), results.end(), 0)) / num_m;
		h_exp[i] = h_exp[i] / num_m;
	//	cout << "finish calculating " << i << "node's expectation" << endl;

		
	}

	hiprandDestroyGenerator(prng);
	
	cout << "Elapsed Time: " << sdkGetTimerValue(&hTimer) + generate_random_number_time * num_n << "ms" << endl;
	cout << "average elapsed time for one variable: " << sdkGetTimerValue(&hTimer) / num_n  + generate_random_number_time<< "ms" << endl;
	cout << "minimal elapsed time for one variable��" << fastest_elapsed_time + generate_random_number_time << "ms" << endl;
	cout << "maximal elapsed time for one variable��" << slowest_elapsed_time + generate_random_number_time << "ms" << endl;

	float sum = 0;
	//cout << "ans:" << endl;
	for (int i = 0; i < num_n; i++){
	//	cout << h_exp[i] << " ";
		sum += h_exp[i];
	}
	//cout << endl;
	//cout << sum << endl;
	cout << "total error: " << fabs(sum  / num_n - 1) << endl;
	cout << endl;
	*/
}	

void solvePagerankByMCGPU(float alpha, int num_m, int threads_per_block, int num_n, int matrix_size, unsigned int seed, 
	int* &h_matrix_l, int* &h_accu_row_size, int* &out_deg){
	int device_id = gpuGetMaxGflopsDeviceId();
	hipSetDevice(device_id);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device_id);

	cout << "running Metro Carlo method at GPU: ";
	cout << prop.name << endl;

	int *d_matrix_l;
	int *d_accu_row_size;
	int *d_out_deg;

	hipMalloc((void **)&d_matrix_l, matrix_size * sizeof(int));
	hipMalloc((void **)&d_accu_row_size, (num_n + 1) * sizeof(int));
	hipMalloc((void **)&d_out_deg, num_n * sizeof(int));

	hipMemcpy(d_matrix_l, h_matrix_l, matrix_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_accu_row_size, h_accu_row_size, (num_n + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_out_deg, out_deg, num_n * sizeof(int), hipMemcpyHostToDevice);

	simulateMC(device_id, num_n, d_matrix_l, d_accu_row_size, alpha, num_m, seed, threads_per_block, out_deg, d_out_deg);

	hipDeviceReset();
}
